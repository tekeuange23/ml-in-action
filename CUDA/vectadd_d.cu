#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>

/**
 * #define N 1_000_000_000, #crashed due to insuffisiant memory.
 * PROOF:
 * a @float: 32_bits == 4_bytes
 * # @total:  1_000_000_000 * 3 == 3_000_000_000 variables
 * @conclusion: require 12_000_000_000 bytes of global memory
 *                   ==     11_718_750 KB    of global memory
 *                   ==      11_444.09 MB    of global memory
 *                   ==         11.175 GB    of global memory
*/
#define N 10
// #define N 100000000

using namespace std;


// Host Vector Initialization
float* vectInit(float value, int n) {
    size_t size = n * sizeof(float);
    float* vect = (float*)malloc(size * sizeof(float));

    if (vect == NULL) {
        printf("Memory allocation failed!\n");
        exit(EXIT_FAILURE);
    } else {
        int i;
        for (i = 0; i < n; i++)
            vect[i] = i * value;
    }

    return vect;
}

// Print the array elements
void vectPrint(float* vect, int n) {
    int i;
    printf("-->: ");
    for (i = 0; i < n; i++)
        printf("%f ", vect[i]);
    printf("\n");
}

// Kernel Function:  Each Thread performs one pair wise addition
__global__
void vectAddKernel(float* C, float* A, float* B, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

// Compute vector sum C = A+B
void vectAdd(float* C, float* A, float* B, int n, bool cuda) {
    if (!cuda) {
        int i;
        for (i = 0; i < n; i++)
            C[i] = A[i] + B[i];
        vectPrint(A, N);
        vectPrint(B, N);
        vectPrint(C, N);
        return;
    }

    /** ******************************************** */
    /**         Part_1: Allocate Device Memory       */
    /** ******************************************** */
    size_t size = n * sizeof(float);
    float* d_A, * d_B, * d_C;
    /**
     * copy the object from the @host_memory to the @local_memory
     * and save the value of the pointer to the @local_memory  */
    hipError_t err_A = hipMalloc((void**)&d_A, size);
    hipError_t err_B = hipMalloc((void**)&d_B, size);
    hipError_t err_C = hipMalloc((void**)&d_C, size);
    if (err_A != hipSuccess || err_B != hipSuccess || err_C != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err_A), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    /** ******************************************** */
    /**           Part_2: Kernel launch code         */
    /** ******************************************** */
    vectAddKernel << <ceil(n / 256.0), 256 >> > (d_C, d_A, d_B, N);

    /** ******************************************** */
    /**            Part_3: Copy in Host Memory       */
    /** ******************************************** */
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    vectPrint(A, N);
    vectPrint(B, N);
    vectPrint(C, N);

    /** ******************************************** */
    /**            Final: Free device vectors        */
    /** ******************************************** */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    // get card info
    // getCardInfo();

    // Init
    float* A, * B, * C;
    A = vectInit(1.0, N);
    B = vectInit(0.1, N);
    C = vectInit(0.0, N);

    // Addition on Host
    auto start_time = chrono::high_resolution_clock::now();
    vectAdd(C, A, B, N, false);
    auto end_time = chrono::high_resolution_clock::now();
    chrono::duration<double> host_time = end_time - start_time;
    std::cout << "Time on Host:: " << host_time.count() << " seconds." << endl;

    // Addition on Device
    start_time = chrono::high_resolution_clock::now();
    vectAdd(C, A, B, N, true);
    end_time = chrono::high_resolution_clock::now();
    chrono::duration<double>  device_time = end_time - start_time;
    std::cout << "Time on Device:: " << device_time.count() << " seconds." << endl;
    std::cout << endl << "Conclusion: computation on the Device is "
        << host_time.count() / device_time.count()
        << " Faster than the Host." << endl;

    // Free Host Memory
    free(A);
    free(B);
    free(C);

    return 0;
    // nvcc -ccbin /usr/bin/g++-11 vectadd_d.cu -o vectadd_d && ./vectadd_d
}